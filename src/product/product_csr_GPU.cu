#include "hip/hip_runtime.h"
#include "headers/product_csr.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

//#define MAX_NNZ_PER_WG 6144
#define MAX_NNZ_PER_WG 4096
#define MAX_BLOCK_THREADS 1024
#define MAX_GRID_SIZE 65536
#define WARP_SIZE 32

__device__ double warp_reduce(double val){
    for(int offset=warpSize/2; offset>0; offset/=2){
        val += __shfl_down_sync(0xffffffff,val, offset);
    }
    return val;
}
 
int* calculate_rows_block(int totalRows, int* irp, int* numBlocks){
    int* rowBlocks = (int*)malloc(MAX_GRID_SIZE*sizeof(int));
    rowBlocks[0] = 0; 
    int sum = 0, last_i= 0, ctr=1; 
    for(int i = 1; i < totalRows; i++){
        sum += irp[i]-irp[i-1];
        if(sum == MAX_NNZ_PER_WG){
            last_i = i;
            rowBlocks[ctr++] = i;
            sum = 0;

        }
        else if( sum > MAX_NNZ_PER_WG){
            if(i - last_i > 1){
                rowBlocks[ctr++] = i-1;
                i--;
            }
            else if(i - last_i == 1){
                rowBlocks[ctr++] = i;
            }
            last_i = i; 
            sum = 0;
        } 
    }
    
    //printf("%d %d\n",ctr,totalRows);
    *numBlocks = ctr;
    rowBlocks[ctr++] = totalRows;
    return rowBlocks;
} 

__global__ void sparseDenseMatrixMul(double* as, int* ja, int* irp, int m, double* multivector, int n, int q, double* resultData)
{
    int row_mat = blockIdx.x * blockDim.x + threadIdx.x;
    int col_vec = threadIdx.y;
    

    if (row_mat < m && col_vec < q)
    {

        int start = irp[row_mat];
        int end = irp[row_mat+1];
        double sum = 0;
        
        
        for (int i = start; i < end; i++)
        {
            sum += as[i] * multivector[ja[i] * q + col_vec];
        }
        
        resultData[row_mat * q + col_vec] = sum;
    }
}


__global__ void csrAdaptiveMult(double* as, int* ja, int* irp, double* multivector, int m, int n, int col_multivector, int* rowBlocks, double* resultData){
    __shared__ double vals[MAX_NNZ_PER_WG];
    __shared__ int cols[MAX_NNZ_PER_WG];
    
    int startRow = rowBlocks[blockIdx.x];
    int stopRow = rowBlocks[blockIdx.x+1];
    long int numRows = stopRow - startRow;
    int nnz = irp[stopRow]-irp[startRow];
    if (numRows > 1){
        //CSR-Stream
        //printf("csr stream\n");
        int tid = threadIdx.x; // indice del thread nel blocco
        int localCol;
        //for(int j = 0; j < col_multivector; j++){
        for(int i = tid; i < nnz; i+= blockDim.x){ 
            localCol = irp[startRow]+i;
            vals[i] = as[localCol];
            //vals[i] *= multivector[ja[localCol]*col_multivector+j];
            cols[i] = ja[localCol];
        }
        int firstCol = irp[startRow];
        int localRow = startRow + tid;
        
        __syncthreads();
        for(int j = 0; j < col_multivector; j++){
            while(localRow < stopRow){
                double temp = 0; 
                for(int i = irp[localRow]-firstCol; i < irp[localRow+1]-firstCol; i++){
                    temp += vals[i]*multivector[cols[i]*col_multivector+j];
                }
                resultData[localRow*col_multivector +j] = temp;
                localRow += blockDim.x;
            }
        }    
        __syncthreads();
        //}
    }else {
        //CSR-Vector
        int threadId = threadIdx.x; // global thread index
        //printf("csr vector\n");
        int warpId = threadId / 32; // Global warp index
        int lane = threadId &(32-1); // thread index within the warp
        //one warp per row
        double val; 
        int col;
        double sum[64] = {0};
        //Questo blocco fa solo questa riga
        if(warpId == 0){            
            for(int i = irp[startRow] + lane; i < irp[startRow+1]; i +=32){
                val = as[i];
                col = ja[i];
                for(int j = 0; j < col_multivector; j++){
                    sum[j] += val*multivector[col*col_multivector + j];
                }    
            }
        }
        for(int i = 0; i < col_multivector; i++){
            sum[i] = warp_reduce(sum[i]);
            if(lane == 0 && warpId == 0){
                
                resultData[startRow*col_multivector + i] = sum[i];   
            }    
        }       
    }
}

__global__ void csrAdaptiveMultOttimizzato(double* as, int* ja, int* irp, double* multivector, int m, int n, int col_multivector, int* rowBlocks, double* resultData){
    __shared__ double vals[MAX_NNZ_PER_WG];
    __shared__ int cols[MAX_NNZ_PER_WG];
    
    int startRow = rowBlocks[blockIdx.x];
    int stopRow = rowBlocks[blockIdx.x+1];
    long int numRows = stopRow - startRow;
    int nnz = irp[stopRow]-irp[startRow];
    int tid = threadIdx.x; // indice del thread nel blocco
    if (numRows > 1){
        //CSR-Stream
        //printf("csr stream\n");
        
        int localCol;
        
        for(int i = tid; i < nnz; i+= blockDim.x){ 
            localCol = irp[startRow]+i;
            vals[i] = as[localCol];
            //vals[i] *= multivector[ja[localCol]*col_multivector+j];
            cols[i] = ja[localCol];
        }
        int firstCol = irp[startRow];
        
        __syncthreads();
        for(int t = tid; t < numRows*col_multivector; t += blockDim.x){
            int localRow = startRow + t/col_multivector;
            int j = t%col_multivector;
            double temp = 0; 
            for(int i = irp[localRow]-firstCol; i < irp[localRow+1]-firstCol; i++){
                temp += vals[i]*multivector[cols[i]*col_multivector + j];
            }
            resultData[localRow*col_multivector +j] = temp;
        }
           
        __syncthreads();    
        
    }else {
        //CSR-Vector
        //printf("csr vector\n");
        int warpId = tid / 32; // Global warp index
        int lane = tid &(32-1); // thread index within the warp
        //one warp per row
        double val; 
        int col;
        double sum[64] = {0};   
        if(nnz < 4096){
            int localCol;
            for(int i = tid; i < nnz; i+= blockDim.x){ 
                localCol = irp[startRow]+i;
                vals[i] = as[localCol];
                cols[i] = ja[localCol];
            }
        }
        __syncthreads();
        if(warpId < col_multivector){
            for(int col_m = warpId; col_m < col_multivector; col_m +=32){
                for(int i = irp[startRow] + lane; i < irp[startRow+1]; i +=32){
                    if(nnz < 4096){
                        val = as[i];
                        col = ja[i];
                    }else{
                        val = as[i];
                        col = ja[i];
                    }
                    sum[col_m] += val*multivector[col*col_multivector + col_m];     
                }
                sum[col_m] = warp_reduce(sum[col_m]);
                if(lane == 0){
                    resultData[startRow*col_multivector + col_m] = sum[col_m];   
                }
            }
        }
    }
}

performance calcola_prodotto_csr_cuda(csr_matrix mat, matrix multivector, matrix* result){
    double* d_as;
    int* d_ja;
    int* d_irp, *d_rowBlocks;
    double* d_multivector, *d_multivector2;
    double* d_result, *d_result2;
    int num_blocks;
    int* res = calculate_rows_block(mat.m, mat.irp,&num_blocks);

    // resetto il device
    checkCudaErrors(hipDeviceReset());

    // Definizione della griglia, del blocco
    int blockX = MAX_BLOCK_THREADS;
    int gridX = num_blocks;
    
    dim3 blockSize(blockX);
    dim3 gridSize(gridX);

    // Alloco
    checkCudaErrors(hipMalloc((void**)&d_as, mat.nz * sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&d_ja, mat.nz * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&d_irp, (mat.m + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&d_rowBlocks, (num_blocks+1) * sizeof(int)));

    checkCudaErrors(hipMalloc((void**)&d_multivector, multivector.m * multivector.n * sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&d_result, mat.m * multivector.n * sizeof(double)));


    // Copia dati sulla GPU
    checkCudaErrors(hipMemcpy(d_as, mat.as, mat.nz * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_ja, mat.ja, mat.nz * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_irp, mat.irp, (mat.m + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_rowBlocks, res, (num_blocks+1) * sizeof(int), hipMemcpyHostToDevice));
    
    checkCudaErrors(hipMemcpy(d_multivector, multivector.coeff, multivector.m * multivector.n * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset(d_result, 0, mat.m * multivector.n * sizeof(double)));


    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, 0));
    
    // Esecuzione del kernel
    csrAdaptiveMultOttimizzato<<<gridSize, blockSize>>>(d_as, d_ja, d_irp, d_multivector, mat.m,mat.n, multivector.n, d_rowBlocks,d_result);

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    //sparseDenseMatrixMul<<<gridSize, blockSize>>>(d_as, d_ja, d_irp, mat.m, d_multivector, mat.n, multivector.n, d_result);
    float time;
    checkCudaErrors(hipEventElapsedTime(&time, start, stop));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    // Copia risultato dalla GPU alla CPU
    double * resultData =(double*) malloc(mat.m * multivector.n * sizeof(double));
    
    checkCudaErrors(hipMemcpy(resultData, d_result, mat.m * multivector.n * sizeof(double), hipMemcpyDeviceToHost));
    
    result->coeff = resultData;
    // Deallocazione memoria
    checkCudaErrors(hipFree(d_ja));
    checkCudaErrors(hipFree(d_as));
    checkCudaErrors(hipFree(d_irp));
    checkCudaErrors(hipFree(d_multivector));
    checkCudaErrors(hipFree(d_result));
    checkCudaErrors(hipFree(d_rowBlocks));

    performance perf;
    perf.time = (double)time/1000;
    perf.bandwidth = (double)(8*(long)(mat.m * multivector.n + mat.nz+multivector.m * multivector.n))+4*(long)(mat.nz +mat.m +1 + num_blocks)/(perf.time);
    perf.bandwidth = perf.bandwidth/pow(10,9);
    return perf;
    
}